#include "hip/hip_runtime.h"
#define CUDA
#include <hip/hip_runtime.h>
#include "devarea.hpp"
#include "protocol.hpp"
#include "debug.h"
#include "ptx_stub.h"

__device__ DeviceArea DEVICE_AREA_GLOBAL_NAME;

extern "C" __global__ void INIT_FUNCTION_NAME()
{
    DEVICE_AREA_GLOBAL_NAME.init();
    __threadfence_system();
}

extern "C" __device__ __attribute__((noinline)) void STORE_OP_FUNCTION_NAME(bool op)
{
    DEVICE_AREA_GLOBAL_NAME.inc(blockIdx.x % 64, op);
}


extern "C" __global__ void force_function_linking(int count)
{
    STORE_OP_FUNCTION_NAME(true);
    STORE_OP_FUNCTION_NAME(false);
}


int main (int argc, char* argv[])
{
    uint64_t* x;
    if(0 != hipMalloc(&x, sizeof(uint64_t)))
    {
        printf("Failed hipMalloc().\n");
        return 1;
    }
    void* buf;
    int buf_size = 64 * 1000;
    if(0 != hipMalloc(&buf, buf_size))
    {
        printf("Failed hipMalloc().\n");
        return 1;
    }
    DeviceArea devarea;

    INIT_FUNCTION_NAME<<<1,1>>>();
    int sync = hipDeviceSynchronize();
    if(sync != 0)
    {
        printf("%s failed, err=%i\n", NAMEOF_INIT_FUNCTION_NAME, sync);
        return 2;
    }

    force_function_linking<<<1,1>>>(10);
    sync = hipDeviceSynchronize();
    if(sync != 0)
    {
        printf("Link function failed, err=%i\n", sync);
        return 2;
    }
    int v = hipMemcpyFromSymbol(&devarea, HIP_SYMBOL(DEVICE_AREA_GLOBAL_NAME), sizeof(DeviceArea), 0, hipMemcpyDeviceToHost);
    if(v!=0)
    {
        printf("Run function failed, err=%i\n", v);
    }
    printf("Value:% i\n", devarea.get_count(1));
    
    printf("PTX stubs tested OK!\n");
    return 0;
}
