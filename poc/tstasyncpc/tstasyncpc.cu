#include "hip/hip_runtime.h"
#include <iostream>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <stdint.h>
#include "hooklib.h"
#include "protocol.hpp"
#include "slimfast.hpp"
#include "consumer.hpp"
#include "impl.hpp"
#include "devlogger.hpp"

static const int NUM_BLOCKS = 2;
static const int NUM_THREADS = 2;
static const int NUM_TOTAL = NUM_BLOCKS * NUM_THREADS;

static const int NUM_VALUES = 3;

__device__ unsigned int counter = 0;
__global__ void producer(unsigned int* data) 
{
    int id = ((blockDim.x * blockIdx.x) + threadIdx.x);
    for(int i = 0; i < NUM_VALUES; ++i )
    {
        data[id] += 1;
        __store_op(&data[id], OP_READ);
        __store_op(&data[id], OP_WRITE);
    }
}

///  host code
    
int main(int argc, char* argv[]) 
{

    Impl impl;

    // Launch the kernel.
    unsigned int* dev_data;
    checkCudaErrors(hipMalloc(&dev_data, sizeof(unsigned int) * NUM_TOTAL));
    checkCudaErrors(hipMemset(dev_data, 0, sizeof(unsigned int) * NUM_TOTAL));
    unsigned int* host_data = (unsigned int*)malloc(sizeof(unsigned int) * NUM_TOTAL);
    
    producer<<<NUM_BLOCKS, NUM_THREADS>>>(dev_data);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(host_data, dev_data, sizeof(unsigned int) * NUM_TOTAL, hipMemcpyDeviceToHost));
    for(int i = 0; i < NUM_TOTAL; ++ i)
    {
        if(host_data[i] != NUM_VALUES)
        {
            fprintf(stderr, "Error at index: %i\n", i);
            exit(-1);
        }
    }

    return 0;
}

