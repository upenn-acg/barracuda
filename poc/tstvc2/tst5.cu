#include "hip/hip_runtime.h"
#include <iostream>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <stdint.h>

static const int NUM_BLOCKS = 1;
static const int NUM_THREADS = 33;
static const int NUM_TOTAL = NUM_BLOCKS * NUM_THREADS;

__global__ void tstfun(volatile unsigned int* data, int repeats) 
{
    int id = ((blockDim.x * blockIdx.x) + threadIdx.x);

    if(blockIdx.x == 0 && threadIdx.x == 0)
        data[0] = 0;
    //__syncthreads();
    if (blockIdx.x == 0 && threadIdx.x == 32)
        data[0] = 1;
}

///  host code
    
int main(int argc, char* argv[]) 
{
    // Launch the kernel.
    unsigned int* dev_data;
    checkCudaErrors(hipMalloc(&dev_data, sizeof(unsigned int) * NUM_TOTAL));
    checkCudaErrors(hipMemset(dev_data, 0, sizeof(unsigned int) * NUM_TOTAL));
    unsigned int* host_data = (unsigned int*)malloc(sizeof(unsigned int) * NUM_TOTAL);
    
    tstfun<<<NUM_BLOCKS, NUM_THREADS>>>(dev_data, 5);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(host_data, dev_data, sizeof(unsigned int) * NUM_TOTAL, hipMemcpyDeviceToHost));
    printf("Success.\n");

    return 0;
}

